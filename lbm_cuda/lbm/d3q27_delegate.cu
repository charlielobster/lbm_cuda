#include <helper_gl.h>
#include <cuda_gl_interop.h>
#include <>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "d3q27_delegate.h"
#include "d3q27_global.cuh"

void d3q27_delegate::launchKernels(lbm_render_mode mode)
{

}

void d3q27_delegate::resetLattice(GLuint pbo)
{

}

void d3q27_delegate::freeCUDA()
{
	hipFree(d3q27_gpu);
	hipFree(array1_gpu);
	hipFree(array2_gpu);
	hipFree(barrier_gpu);
	hipGraphicsUnregisterResource(cuda_pbo_resource);
}

