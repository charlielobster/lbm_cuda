#include "hip/hip_runtime.h"
#include <helper_gl.h>
#include <cuda_gl_interop.h>
#include <>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "lbm_delegate.cuh"
#include "lbm_global.cuh"

void lbm_delegate::printDeviceInfo()
{
	int nDevices = 0;
	hipError_t ce = hipGetDeviceCount(&nDevices);
	hipDeviceProp_t prop;
	for (int i = 0; i < nDevices; ++i)
	{
		ce = hipGetDeviceProperties(&prop, i);
		printf("Device number: %d\n", i);
		printf("Device name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max threads in X-dimension of block: %d\n", prop.maxThreadsDim[0]);
		printf("Max threads in Y-dimension of block: %d\n", prop.maxThreadsDim[1]);
		printf("Max threads in Z-dimension of block: %d\n\n", prop.maxThreadsDim[2]);
		if (ce != hipSuccess) { printf("error: %s\n", hipGetErrorString(ce)); }
	}
}

void lbm_delegate::initPboResource(GLuint pbo)
{
	hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);
}

void lbm_delegate::initCUDA(d2q9_node* d2q9, lbm_node* array1, lbm_node* array2, unsigned char* barrier)
{
	hipError_t ce = hipMalloc(&d2q9_gpu, 9 * sizeof(d2q9_node));
	ce = hipMalloc(&barrier_gpu, sizeof(unsigned char) * LATTICE_DIMENSION);
	ce = hipMalloc(&array1_gpu, sizeof(lbm_node) * LATTICE_DIMENSION);
	ce = hipMalloc(&array2_gpu, sizeof(lbm_node) * LATTICE_DIMENSION);

	ce = hipMemcpy(d2q9_gpu, d2q9, sizeof(d2q9_node) * 9, hipMemcpyHostToDevice);
	ce = hipMemcpy(barrier_gpu, barrier, sizeof(unsigned char) * LATTICE_DIMENSION, hipMemcpyHostToDevice);
	ce = hipMemcpy(array1_gpu, array1, sizeof(lbm_node) * LATTICE_DIMENSION, hipMemcpyHostToDevice);
	ce = hipMemcpy(array2_gpu, array2, sizeof(lbm_node) * LATTICE_DIMENSION, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
}

void lbm_delegate::launchKernels(render_mode mode, bool barriersUpdated, unsigned char* barrier)
{
	//reset image pointer
	uchar4* d_out = 0;

	//set d_out as a texture memory pointer
	hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
	hipGraphicsResourceGetMappedPointer((void**)&d_out, NULL, cuda_pbo_resource);

	//launch cuda kernels to calculate LBM step
	for (int i = 0; i < STEPS_PER_RENDER; i++)
	{
		if (barriersUpdated)
		{
			hipMemcpy(barrier_gpu, barrier, sizeof(unsigned char) * LATTICE_DIMENSION, hipMemcpyHostToDevice);
			hipDeviceSynchronize(); // Wait for the GPU to finish
		}

		//determine number of threads and blocks required
		dim3 threads_per_block = dim3(32, 32, 1);
		dim3 number_of_blocks = dim3(LATTICE_WIDTH / 32 + 1, LATTICE_HEIGHT / 32 + 1, 1);

		collide<<<number_of_blocks, threads_per_block>>>(d2q9_gpu, array1_gpu, array2_gpu, barrier_gpu);
		hipDeviceSynchronize();

		stream<<<number_of_blocks, threads_per_block>>>(d2q9_gpu, array2_gpu, array1_gpu, barrier_gpu);
		hipDeviceSynchronize();

		bounce<<<number_of_blocks, threads_per_block>>>(d2q9_gpu, array2_gpu, array1_gpu, barrier_gpu, d_out);
		hipDeviceSynchronize();

		color<<<number_of_blocks, threads_per_block>>>(mode, array1_gpu, d_out, barrier);
		hipDeviceSynchronize();
	}

	//unmap the resources for next time
	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
}

void lbm_delegate::freeCUDA()
{
	hipFree(d2q9_gpu);
	hipFree(array1_gpu);
	hipFree(array2_gpu);
	hipFree(barrier_gpu);
	hipGraphicsUnregisterResource(cuda_pbo_resource);
}
